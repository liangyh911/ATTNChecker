#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <cstdint>
#include <stdlib.h>
#include <iostream>
#include <ATen/ATen.h>

template <typename T>
__global__ void
detect_correct_col(T * dA, int64_t ldda, T E, int64_t stridea,
						     T * dA_colchk, 	int64_t ldda_colchk,	int64_t stride_colchk,
						     T * dA_colchk_r, int64_t ldda_colchk_r,	int64_t stride_colchk_r){
    //printf("col_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
	dA_colchk = dA_colchk + blockIdx.x * stride_colchk;
	dA_colchk_r = dA_colchk_r + blockIdx.x * stride_colchk_r;
    
    //determine the specific colum to process
	// printf("determin the specific colum to process. \n");
    dA = dA + threadIdx.x * ldda;
    dA_colchk   = dA_colchk   + threadIdx.x * ldda_colchk;
    dA_colchk_r = dA_colchk_r + threadIdx.x * ldda_colchk_r;
	
    T d1 = (*dA_colchk)       - (*dA_colchk_r);
    T d2 = (*(dA_colchk + 1)) - (*(dA_colchk_r + 1));
	
    //error detected
	// printf("error detected. \n");
    if(fabs(d1) > E) {
    	//locate the error
		int loc = round(d2 / d1) - 1;
		printf("[col check]error detected (d1 = %.6f, d2 = %.6f, loc = %d) \n", (float)d1, (float)d2, loc);
			
		//the sum of the rest correct number except the error one
		T sum = 0.0;
		for(int i = 0; i < ldda; i++) {
			if (i != loc) {
				sum +=	*(dA + i); 
			}
		}
		//correct the error
		*(dA + loc) = *dA_colchk - sum;
    }
}

template<typename T>
__global__ void
detect_correct_row(T * dA, int64_t ldda, T E, int64_t stridea,
						     T * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
						     T * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r){
    // printf("row_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
    dA_rowchk = dA_rowchk + blockIdx.x * stride_rowchk;
    dA_rowchk_r = dA_rowchk_r + blockIdx.x * stride_rowchk_r;
        
    //determine the specific row to process
	// printf("determin the specific row to process. \n");
	dA = dA + threadIdx.x;
    dA_rowchk   = dA_rowchk   + threadIdx.x;
    dA_rowchk_r = dA_rowchk_r + threadIdx.x;
	
    T d1 = (*dA_rowchk)                 - (*dA_rowchk_r);
    T d2 = (*(dA_rowchk + ldda_rowchk)) - (*(dA_rowchk_r + ldda_rowchk_r));
	
    //error detected
	// printf("error detected. \n");
    if(fabs(d1) > E) {
		//locate the error
		int loc = round(d2 / d1) - 1;
		printf("[row check]error detected (d1 = %.6f, d2 = %.6f, loc = %d) \n", (float)d1, (float)d2, loc);
			
		//the sum of the rest correct number except the error one
		T sum = 0.0;
		for (int i = 0; i < ldda; i++) {
		    if (i != loc) {
				sum +=	*(dA + i * ldda); 
		    }
		}
        //correct the error
		*(dA + loc * ldda) = *dA_rowchk - sum;
     }
}
// void detect_correct_row_lancher(float * dA, int64_t ldda, float E, int64_t stridea,
// 						     float * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
// 						     float * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r,
// 							 int numBlock, int64_t numThread, hipStream_t stream_rowchk){
	
// 	detect_correct_row<<<dim3(numBlock), dim3(numThread), 0, stream_rowchk>>>(dA, ldda, E, stridea,
//                                           dA_rowchk, ldda_rowchk,     stride_rowchk,
//                                           dA_rowchk_r, ldda_rowchk_r, stride_rowchk_r);
// }

__global__ void
detect_correct_col(at::Half * dA, int64_t ldda, at::Half E, int64_t stridea,
						     at::Half * dA_colchk, 	int64_t ldda_colchk,	int64_t stride_colchk,
						     at::Half * dA_colchk_r, int64_t ldda_colchk_r,	int64_t stride_colchk_r){
    //printf("col_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
	dA_colchk = dA_colchk + blockIdx.x * stride_colchk;
	dA_colchk_r = dA_colchk_r + blockIdx.x * stride_colchk_r;
    
    //determine the specific colum to process
	// printf("determin the specific colum to process. \n");
    dA = dA + threadIdx.x * ldda;
    dA_colchk   = dA_colchk   + threadIdx.x * ldda_colchk;
    dA_colchk_r = dA_colchk_r + threadIdx.x * ldda_colchk_r;
	
    float d1 = (float)((*dA_colchk)       - (*dA_colchk_r));
    float d2 = (float)((*(dA_colchk + 1)) - (*(dA_colchk_r + 1)));
	
    //error detected
	// printf("error detected. \n");
    if(fabs(d1) > E) {
    	//locate the error
		// int loc = __half2int_rn(d2 / d1) - 1;
		int loc = round(d2 / d1) - 1;
		printf("[col check]error detected (val1 = %.6f, val2 = %.6f), (d1 = %.6f, d2 = %.6f, loc = %d) \n", \
												(float)(*dA_colchk), (float)(*dA_colchk_r), (float)(d1), (float)(d2), loc);
			
		//the sum of the rest correct number except the error one
		at::Half sum = 0;
		for(int i = 0; i < ldda; i++) {
			if (i != loc) {
				sum = sum + (*(dA + i)); 
			}
		}
		//correct the error
		*(dA + loc) = *dA_colchk - sum;
    }
}
// void detect_correct_col_lancher(at::Half * dA, int64_t ldda, float E, int64_t stridea,
// 						     at::Half * dA_colchk, 	int64_t ldda_colchk,	int64_t stride_colchk,
// 						     at::Half * dA_colchk_r, int64_t ldda_colchk_r,	int64_t stride_colchk_r,
// 							 int numBlock, int64_t numThread, hipStream_t stream_colchk){
	
// 	detect_correct_col<<<dim3(numBlock), dim3(numThread), 0, stream_colchk>>>(dA, ldda, E, stridea,
//                                             dA_colchk,      ldda_colchk,    stride_colchk,
//                                             dA_colchk_r,    ldda_colchk_r,  stride_colchk_r);
// }

__global__ void
detect_correct_row(at::Half * dA, int64_t ldda, at::Half E, int64_t stridea,
						     at::Half * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
						     at::Half * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r){
    // printf("row_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
    dA_rowchk = dA_rowchk + blockIdx.x * stride_rowchk;
    dA_rowchk_r = dA_rowchk_r + blockIdx.x * stride_rowchk_r;
        
    //determine the specific row to process
	// printf("determin the specific row to process. \n");
	dA = dA + threadIdx.x;
    dA_rowchk   = dA_rowchk   + threadIdx.x;
    dA_rowchk_r = dA_rowchk_r + threadIdx.x;
	
    float d1 =  (float)((*dA_rowchk)                 - (*dA_rowchk_r));
    float d2 =  (float)((*(dA_rowchk + ldda_rowchk)) - (*(dA_rowchk_r + ldda_rowchk_r)));
	
    //error detected
	// printf("error detected. \n");
    if(float(d1) > E) {
		//locate the error
		// int loc = __half2int_rn(d2 / d1) - 1;
		int loc = round(d2 / d1) - 1;
		printf("[row check]error detected (val1 = %.6f, val2 = %.6f), (d1 = %.6f, d2 = %.6f, loc = %d) \n", \
												(float)(*dA_rowchk), (float)(*dA_rowchk_r), (float)(d1), (float)(d2), loc);
			
		//the sum of the rest correct number except the error one
		at::Half sum = 0.0;
		for (int i = 0; i < ldda; i++) {
		    if (i != loc) {
				sum += *(dA + i * ldda); 
		    }
		}
        //correct the error
		*(dA + loc * ldda) = *dA_rowchk - sum;
     }
}
// void detect_correct_row_lancher(at::Half * dA, int64_t ldda, float E, int64_t stridea,
// 						     at::Half * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
// 						     at::Half * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r,
// 							 int numBlock, int64_t numThread, hipStream_t stream_rowchk){
	
// 	detect_correct_row<<<dim3(numBlock), dim3(numThread), 0, stream_rowchk>>>(dA, ldda, E, stridea,
//                                           dA_rowchk, ldda_rowchk,     stride_rowchk,
//                                           dA_rowchk_r, ldda_rowchk_r, stride_rowchk_r);
// }