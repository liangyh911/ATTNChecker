#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <cstdint>
#include <stdlib.h>
#include <iostream>

#include <ATen/ATen.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/CUDADataType.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAFunctions.h>
#include <c10/macros/Export.h>
#include <c10/util/irange.h>

/*
__constant__ float CHK_V_A[2*72];

__global__ void encode_col_v1(int m, int k, int num_batches,
					float *dA, int64_t ldda, int64_t strideA, 
                     float *chk_v, int64_t ld_chk_v, int64_t stride_chk_v,
					 float *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk) {

	const int batch_id = blockIdx.x;
	const int x = threadIdx.x;
	const int y = threadIdx.y;
	dA = dA + batch_id * strideA;
	chk_v = chk_v + batch_id * stride_chk_v;
	dA_colchk = dA_colchk + batch_id * strideA_colchk;

	float res = 0.0;
	for (int i = 0; i < m; i++) {
		res += chk_v[y + i * ld_chk_v] * dA[i + x * ldda];
	}
	dA_colchk[y + x * ldda_colchk] = res;
}

__global__ void encode_col_v2(int m, int k, int num_batches,
					float *dA, int64_t ldda, int64_t strideA, 
                     float *chk_v, int64_t ld_chk_v, int64_t stride_chk_v,
					 float *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk) {

	const int batch_id = blockIdx.x;
	const int x = threadIdx.x;
	const int y = threadIdx.y;
	dA = dA + batch_id * strideA;
	// chk_v = chk_v + batch_id * stride_chk_v;
	dA_colchk = dA_colchk + batch_id * strideA_colchk;

	float res = 0.0;
	for (int i = 0; i < m; i++) {
		res += CHK_V_A[y + i * ld_chk_v] * dA[i + x * ldda];
	}
	dA_colchk[y + x * ldda_colchk] = res;
}

__global__ void encode_col_v3(int m, int k, int num_batches,
					float *dA, int64_t ldda, int64_t strideA, 
                     float *chk_v, int64_t ld_chk_v, int64_t stride_chk_v,
					 float *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk) {

	extern __shared__ float dA_sm [];
	int ldda_sm = m; 

	const int batch_id = blockIdx.x;
	const int x = threadIdx.x;
	const int y = threadIdx.y;
	dA = dA + batch_id * strideA;
	chk_v = chk_v + batch_id * stride_chk_v;
	dA_colchk = dA_colchk + batch_id * strideA_colchk;

	// if (y == 0)	{
		for (int i = 0; i < k/2; i++) {
			dA_sm[x + (i+y*k/2) * ldda_sm] = dA[x + (i+y*k/2) * ldda];
		}
	// }
	__syncthreads();

	if (x < k) {
		float res = 0.0;
		for (int i = 0; i < m; i++) {
			// if (x == 0 && y == 0)
			// printf("%f %f %f\n", chk_v[y + i * ld_chk_v], dA_sm[i + x * ldda_sm], res);
			res += chk_v[y + i * ld_chk_v] * dA_sm[i + x * ldda_sm];
		}
		dA_colchk[y + x * ldda_colchk] = res;
	}
}

__global__ void encode_col_v4(int m, int k, int num_batches,
					float *dA, int64_t ldda, int64_t strideA, 
                     float *chk_v, int64_t ld_chk_v, int64_t stride_chk_v,
					 float *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk) {

	extern __shared__ float dA_sm [];
	int ldda_sm = m; 

	const int batch_id = blockIdx.x;
	const int x = threadIdx.x;
	const int y = threadIdx.y;
	dA = dA + batch_id * strideA;
	chk_v = chk_v + batch_id * stride_chk_v;
	dA_colchk = dA_colchk + batch_id * strideA_colchk;

	for (int i = 0; i < k/2; i++) {
		dA_sm[x + (i+y*k/2) * ldda_sm] = dA[x + (i+y*k/2) * ldda];
	}

	__syncthreads();

	if (x < k) {
		float res = 0.0;
		for (int i = 0; i < m; i++) {
			// if (x == 0 && y == 0)
			// printf("%f %f %f\n", chk_v[y + i * ld_chk_v], dA_sm[i + x * ldda_sm], res);
			res += 1 * dA_sm[i + x * ldda_sm];
		}
		dA_colchk[y + x * ldda_colchk] = res;
	}
}
*/
template<typename T, int NROW, int NCOL, int C>
__global__ void encode_col_v5(int num_batches,
					T *dA, int64_t ldda, int64_t strideA, 
					 T *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk) {

	extern __shared__ T dA_sm [];

	const int batch_id = blockIdx.x;
	const int tid = threadIdx.x;
	const int y_load = tid / NROW;
	const int x_load = tid % NROW;
	const int y_compute = tid / NCOL;
	const int x_compute = tid % NCOL;
	dA = dA + batch_id * strideA;
	dA_colchk = dA_colchk + batch_id * strideA_colchk;

	for (int i = 0; i < NCOL; i += C) {
		dA_sm[x_load+(NROW+1)*(i+y_load)] = dA[x_load+(NROW)*(i+y_load)];
	}	
	__syncthreads();

	if (x_compute < NCOL && y_compute < 2) {
		T res = 0.0;
		T * dA_col = &dA_sm[x_compute * (NROW+1)];
		if (y_compute == 0) {
			for (int i = 0; i < NROW; i++) {
				res += dA_col[i];
			}
		}
		if (y_compute == 1) {
			for (int i = 0; i < NROW; i++) {
				res += (T)(i+1) * dA_col[i];
			}
		}
		dA_colchk[y_compute + x_compute * ldda_colchk] = res;
	}
}
template<typename T, int NROW, int NCOL, int C>
void encode_col_lancher(int num_batches,
					T *dA, int64_t ldda, int64_t strideA, 
					T *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk,
					hipStream_t stream_colchk){
	
	encode_col_v5<T, NROW, NCOL, C><<<num_batches, dim3(NROW*4, 1), (NROW+1)*NCOL*sizeof(T), stream_colchk>>>(num_batches,
                  dA, ldda, strideA, 
                  dA_colchk, ldda_colchk, strideA_colchk);
}

template<typename T, int NROW, int NCOL>
__global__ void encode_row_v5(int num_batches,
					T *dA, int64_t ldda, int64_t strideA, 
					 T *dA_rowchk, int64_t ldda_rowchk, int64_t strideA_rowchk) {

	const int batch_id = blockIdx.x;
	const int tid = threadIdx.x;
	const int y = tid / NROW;
	const int x = tid % NROW;
	dA = dA + batch_id * strideA;
	dA_rowchk = dA_rowchk + batch_id * strideA_rowchk;

	// printf("%d %d\n", x, y);

	if (x < NROW && y < 2) {
		T res = 0.0;
		T * dA_row = &dA[x];
		if (y == 0) {
			for (int i = 0; i < NCOL; i++) {
				res += dA_row[i * NROW];
			}
		}
		if (y == 1) {
			for (int i = 0; i < NCOL; i++) {
				res += (T)(i+1) * dA_row[i * NROW];
			}
		}
		dA_rowchk[y * NROW + x] = res;
	}
}
template<typename T, int NROW, int NCOL>
void encode_row_lancher(int num_batches,
					T *dA, int64_t ldda, int64_t strideA, 
					 T *dA_rowchk, int64_t ldda_rowchk, int64_t strideA_rowchk,
					 hipStream_t stream_rowchk){
	
	encode_row_v5<T, NROW, NCOL><<<num_batches, dim3(NROW*2, 1, 1), 0, stream_rowchk>>>(num_batches,
                  dA, ldda, strideA, 
                  dA_rowchk, ldda_rowchk, strideA_rowchk);
}

template<typename T, int NROW, int NCOL, int C>
__global__ void update_col_v5(int num_batches,
					T *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk, 
					T *dB, int64_t lddb, int64_t strideB, 
					T *dC_colchk, int64_t lddc_colchk, int64_t strideC_colchk) {

	extern __shared__ T sm [];
	T * dA_colchk_sm = sm;
	T * dB_sm = sm + 2*NROW;

	const int batch_id = blockIdx.x;
	const int tid = threadIdx.x;
	int y_load = tid / NROW;
	int x_load = tid % NROW;

	const int y_compute = tid / NCOL;
	const int x_compute = tid % NCOL;
	dA_colchk = dA_colchk + batch_id * strideA_colchk;
	dB = dB + batch_id * strideB;
	dC_colchk = dC_colchk + batch_id * strideC_colchk;

	if (tid < 2*NROW) {
		dA_colchk_sm[tid] = dA_colchk[tid];
	}
	for (int i = 0; i < NCOL; i += C) {
		dB_sm[x_load+(NROW+1)*(i+y_load)] = dB[x_load+(NROW)*(i+y_load)];
	}	
	__syncthreads();

	// printf("%d %d\n", x, y);

	if (x_compute < NCOL && y_compute < 2) {
		T res = 0.0;
		T * row = &dA_colchk_sm[y_compute];
		// T * row = &dA_colchk[y_compute];
		T * col = &dB_sm[x_compute * (NROW+1)];
		for (int i = 0; i < NROW; i++) {
			res += row[i * 2] * col[i];
			// res += 1 * col[i];
		}

		dC_colchk[y_compute + x_compute * 2] = res;
	}
}
template<typename T, int NROW, int NCOL, int C>
void update_col_lancher(int num_batches,
					T *dA_colchk, int64_t ldda_colchk, int64_t strideA_colchk, 
					T *dB, int64_t lddb, int64_t strideB, 
					T *dC_colchk, int64_t lddc_colchk, int64_t strideC_colchk,
					hipStream_t stream_colchk){
	
	update_col_v5<T, NROW, NCOL, C><<<num_batches, dim3(NROW*4, 1, 1), ((NROW+1)*NCOL+2*NROW) * sizeof(T), stream_colchk>>>(num_batches,
                    dA_colchk, ldda_colchk, strideA_colchk, 
                    dB, lddb, strideB, 
                    dC_colchk, lddc_colchk, strideC_colchk);
}


template<typename T, int NROW, int NCOL>
__global__ void update_row_v5(int num_batches,
					T *dA, int64_t ldda, int64_t strideA, 
					T *dB_rowchk, int64_t lddb_rowchk, int64_t strideB_rowchk,
					T *dC_rowchk, int64_t lddc_rowchk, int64_t strideC_rowchk) {

	// extern __shared__ T dB_rowchk [];

	const int batch_id = blockIdx.x;
	const int tid = threadIdx.x;
	const int y = tid / NROW;
	const int x = tid % NROW;
	dA = dA + batch_id * strideA;
	dB_rowchk = dB_rowchk + batch_id * strideB_rowchk;
	dC_rowchk = dC_rowchk + batch_id * strideC_rowchk;


	// printf("%d %d\n", x, y);

	if (x < NROW && y < 2) {
		T res = 0.0;
		T * row = &dA[x];
		T * col = &dB_rowchk[y*NCOL];
		for (int i = 0; i < NCOL; i++) {
			res += col[i] * row[i * NROW];
		}
		dC_rowchk[y * NROW + x] = res;
	}
}
template<typename T, int NROW, int NCOL>
void update_row_lancher(int num_batches,
					T *dA, int64_t ldda, int64_t strideA, 
					T *dB_rowchk, int64_t lddb_rowchk, int64_t strideB_rowchk,
					T *dC_rowchk, int64_t lddc_rowchk, int64_t strideC_rowchk,
					hipStream_t stream_rowchk){
	
	update_row_v5<T, NROW, NCOL><<<num_batches, dim3(NROW*2, 1, 1), (2*NCOL) * sizeof(T), stream_rowchk>>>(num_batches,
                    dA, ldda, strideA, 
                    dB_rowchk, lddb_rowchk, strideB_rowchk, 
                    dC_rowchk, lddc_rowchk, strideC_rowchk);
}


__global__ void
detect_correct_col(float * dA, int64_t ldda, float E, int64_t stridea,
						     float * dA_colchk, 	int64_t ldda_colchk,	int64_t stride_colchk,
						     float * dA_colchk_r, int64_t ldda_colchk_r,	int64_t stride_colchk_r){
    //printf("col_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
	dA_colchk = dA_colchk + blockIdx.x * stride_colchk;
	dA_colchk_r = dA_colchk_r + blockIdx.x * stride_colchk_r;
    
    //determine the specific colum to process
	// printf("determin the specific colum to process. \n");
    dA = dA + threadIdx.x * ldda;
    dA_colchk   = dA_colchk   + threadIdx.x * ldda_colchk;
    dA_colchk_r = dA_colchk_r + threadIdx.x * ldda_colchk_r;
	
    float d1 = (*dA_colchk)       - (*dA_colchk_r);
    float d2 = (*(dA_colchk + 1)) - (*(dA_colchk_r + 1));
	
    //error detected
	// printf("error detected. \n");
    if(fabs(d1) > E) {
    	//locate the error
		int loc = round(d2 / d1) - 1;
		printf("[col check]error detected (d1 = %.6f, d2 = %.6f, loc = %d) \n", d1, d2, loc);
			
		//the sum of the rest correct number except the error one
		float sum = 0.0;
		for(int i = 0; i < ldda; i++) {
			if (i != loc) {
				sum +=	*(dA + i); 
			}
		}
		//correct the error
		*(dA + loc) = *dA_colchk - sum;
    }
}
void detect_correct_col_lancher(float * dA, int64_t ldda, float E, int64_t stridea,
						     float * dA_colchk, 	int64_t ldda_colchk,	int64_t stride_colchk,
						     float * dA_colchk_r, int64_t ldda_colchk_r,	int64_t stride_colchk_r,
							 int numBlock, int64_t numThread, hipStream_t stream_colchk){
	
	detect_correct_col<<<dim3(numBlock), dim3(numThread), 0, stream_colchk>>>(dA, ldda, E, stridea,
                                            dA_colchk,      ldda_colchk,    stride_colchk,
                                            dA_colchk_r,    ldda_colchk_r,  stride_colchk_r);
}

__global__ void
detect_correct_row(float * dA, int64_t ldda, float E, int64_t stridea,
						     float * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
						     float * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r){
    // printf("row_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
    dA_rowchk = dA_rowchk + blockIdx.x * stride_rowchk;
    dA_rowchk_r = dA_rowchk_r + blockIdx.x * stride_rowchk_r;
        
    //determine the specific row to process
	// printf("determin the specific row to process. \n");
	dA = dA + threadIdx.x;
    dA_rowchk   = dA_rowchk   + threadIdx.x;
    dA_rowchk_r = dA_rowchk_r + threadIdx.x;
	
    float d1 = (*dA_rowchk)                 - (*dA_rowchk_r);
    float d2 = (*(dA_rowchk + ldda_rowchk)) - (*(dA_rowchk_r + ldda_rowchk_r));
	
    //error detected
	// printf("error detected. \n");
    if(fabs(d1) > E) {
		//locate the error
		int loc = round(d2 / d1) - 1;
		printf("[row check]error detected (d1 = %.6f, d2 = %.6f, loc = %d) \n", d1, d2, loc);
			
		//the sum of the rest correct number except the error one
		float sum = 0.0;
		for (int i = 0; i < ldda; i++) {
		    if (i != loc) {
				sum +=	*(dA + i * ldda); 
		    }
		}
        //correct the error
		*(dA + loc * ldda) = *dA_rowchk - sum;
     }
}
void detect_correct_row_lancher(float * dA, int64_t ldda, float E, int64_t stridea,
						     float * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
						     float * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r,
							 int numBlock, int64_t numThread, hipStream_t stream_rowchk){
	
	detect_correct_row<<<dim3(numBlock), dim3(numThread), 0, stream_rowchk>>>(dA, ldda, E, stridea,
                                          dA_rowchk, ldda_rowchk,     stride_rowchk,
                                          dA_rowchk_r, ldda_rowchk_r, stride_rowchk_r);
}

__global__ void
detect_correct_col(at::Half * dA, int64_t ldda, at::Half E, int64_t stridea,
						     at::Half * dA_colchk, 	int64_t ldda_colchk,	int64_t stride_colchk,
						     at::Half * dA_colchk_r, int64_t ldda_colchk_r,	int64_t stride_colchk_r){
    //printf("col_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
	dA_colchk = dA_colchk + blockIdx.x * stride_colchk;
	dA_colchk_r = dA_colchk_r + blockIdx.x * stride_colchk_r;
    
    //determine the specific colum to process
	// printf("determin the specific colum to process. \n");
    dA = dA + threadIdx.x * ldda;
    dA_colchk   = dA_colchk   + threadIdx.x * ldda_colchk;
    dA_colchk_r = dA_colchk_r + threadIdx.x * ldda_colchk_r;
	
    float d1 = (float)((*dA_colchk)       - (*dA_colchk_r));
    float d2 = (float)((*(dA_colchk + 1)) - (*(dA_colchk_r + 1)));
	
    //error detected
	// printf("error detected. \n");
    if(float(d1) > E) {
    	//locate the error
		// int loc = __half2int_rn(d2 / d1) - 1;
		int loc = round(d2 / d1) - 1;
		printf("[col check]error detected (val1 = %.6f, val2 = %.6f), (d1 = %.6f, d2 = %.6f, loc = %d) \n", \
												(float)(*dA_colchk), (float)(*dA_colchk_r), (float)(d1), (float)(d2), loc);
			
		//the sum of the rest correct number except the error one
		at::Half sum = 0;
		for(int i = 0; i < ldda; i++) {
			if (i != loc) {
				sum = sum + (*(dA + i)); 
			}
		}
		//correct the error
		*(dA + loc) = *dA_colchk - sum;
    }
}
void detect_correct_col_lancher(at::Half * dA, int64_t ldda, float E, int64_t stridea,
						     at::Half * dA_colchk, 	int64_t ldda_colchk,	int64_t stride_colchk,
						     at::Half * dA_colchk_r, int64_t ldda_colchk_r,	int64_t stride_colchk_r,
							 int numBlock, int64_t numThread, hipStream_t stream_colchk){
	
	detect_correct_col<<<dim3(numBlock), dim3(numThread), 0, stream_colchk>>>(dA, ldda, E, stridea,
                                            dA_colchk,      ldda_colchk,    stride_colchk,
                                            dA_colchk_r,    ldda_colchk_r,  stride_colchk_r);
}

__global__ void
detect_correct_row(at::Half * dA, int64_t ldda, at::Half E, int64_t stridea,
						     at::Half * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
						     at::Half * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r){
    // printf("row_chk kernel func. \n");
	//determin the block to process
	// printf("determin the block to process. \n");
    dA = dA + blockIdx.x * stridea;
    dA_rowchk = dA_rowchk + blockIdx.x * stride_rowchk;
    dA_rowchk_r = dA_rowchk_r + blockIdx.x * stride_rowchk_r;
        
    //determine the specific row to process
	// printf("determin the specific row to process. \n");
	dA = dA + threadIdx.x;
    dA_rowchk   = dA_rowchk   + threadIdx.x;
    dA_rowchk_r = dA_rowchk_r + threadIdx.x;
	
    float d1 =  (float)((*dA_rowchk)                 - (*dA_rowchk_r));
    float d2 =  (float)((*(dA_rowchk + ldda_rowchk)) - (*(dA_rowchk_r + ldda_rowchk_r)));
	
    //error detected
	// printf("error detected. \n");
    if(float(d1) > E) {
		//locate the error
		// int loc = __half2int_rn(d2 / d1) - 1;
		int loc = round(d2 / d1) - 1;
		printf("[row check]error detected (val1 = %.6f, val2 = %.6f), (d1 = %.6f, d2 = %.6f, loc = %d) \n", \
												(float)(*dA_rowchk), (float)(*dA_rowchk_r), (float)(d1), (float)(d2), loc);
			
		//the sum of the rest correct number except the error one
		at::Half sum = 0.0;
		for (int i = 0; i < ldda; i++) {
		    if (i != loc) {
				sum += *(dA + i * ldda); 
		    }
		}
        //correct the error
		*(dA + loc * ldda) = *dA_rowchk - sum;
     }
}
void detect_correct_row_lancher(at::Half * dA, int64_t ldda, float E, int64_t stridea,
						     at::Half * dA_rowchk, 	int64_t ldda_rowchk,	int64_t stride_rowchk,
						     at::Half * dA_rowchk_r, int64_t ldda_rowchk_r,	int64_t stride_rowchk_r,
							 int numBlock, int64_t numThread, hipStream_t stream_rowchk){
	
	detect_correct_row<<<dim3(numBlock), dim3(numThread), 0, stream_rowchk>>>(dA, ldda, E, stridea,
                                          dA_rowchk, ldda_rowchk,     stride_rowchk,
                                          dA_rowchk_r, ldda_rowchk_r, stride_rowchk_r);
}